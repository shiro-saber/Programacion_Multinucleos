#include <iostream> //ionput output party
#include <opencv2/opencv.hpp> // si dice open cv seguro que son las librerias para hacer operaciones matematicas
#include <stdio.h> // o cielos no era estudio?????
#include "cudaKernel.cu" // MWUWHAHAHAHA vas a tener que revisar otro archivo // que tiene mas comentarios ya sabes por linea

using namespace cv; //si, este es para incluir el namespace cv de opencv
using namespace std; // namespace de Solos Traumados y Dolidos 

int main(int argc, char** argv) //estoy seguro que he visto esto antes epro aun no se que es... tal vez cubells lo sepa
{
    hipEvent_t inicio, fin,st2,fn2; // eventos de cuda??? espero me invviten 
    float tiempo, tt2; // tiempo y mas tiempo = tiempo^2

    int blok, threads; // bloques para contruir y threads para tejer 
    blok = atoi(argv[1]); // الله أكبر
    threads = atoi(argv[2]); // en serio lees todos estos comentarios obvios????

    cout << "bloques: " << blok << "\tthreads: "<< threads << endl; // por si tienes alts heimer y se te olvido cuantos threads y bloques pussite aqui te lo recordamos

    hipEventCreate( &st2 );// sera mi invitacion apra ir al evento de CUDA?
    hipEventCreate( &fn2 ); // si si es!!
    hipEventRecord( st2, 0 ); // que empieze al fiesta 

    IplImage* image; // hmmm... elefantes??
    IplImage* sharmuta;// sip elefantes

    image = cvLoadImage("salon.jpeg", CV_LOAD_IMAGE_GRAYSCALE); // elefantes normales (osea grises) 
    sharmuta = cvLoadImage("salon.jpeg", CV_LOAD_IMAGE_COLOR); // elefantes coloridos (osea estas en drogas)

    if(!image ) // osea que borraste la imgen genio
    {
        cout << "Could not open or find the image" << std::endl; // se kago todo 
        return -1; // :'( 
    }


    IplImage* image2 = cvCreateImage(cvGetSize(image),IPL_DEPTH_32F,image->nChannels); // pues casteamso nuestros elefantes a matrices 
    IplImage* image3 = cvCreateImage(cvGetSize(image),IPL_DEPTH_32F,image->nChannels); // la misma cosa 

    //Convert the input image to float
    cvConvert(image,image3);// tarduccion: que lo convertimos a float 

    float *output = (float*)image2->imageData; // umm si creo que ya lo dije no?
    float *input =  (float*)image3->imageData; // deberias poner mas atencion a los comentarios que pongo 
    
    hipEventCreate( &inicio ); // ViVa La pArI LoKa
    hipEventCreate( &fin ); // todo lo bueno tiene un final... pero esto es el principio del fin. o eso es lo que ahi dice
    hipEventRecord( inicio, 0 );//para medir el tiempo de EJECUCION

    // llamemos al kernel 
    kernelcall(input, output, image->width,image->height, image3->widthStep, blok,threads); // como la unica funcion que importa en todo el codigo
    
    hipEventRecord( fin, 0 );// se acabo 
    hipEventSynchronize( fin ); // wiiiiiiiii
    hipEventElapsedTime( &tiempo, inicio, fin );//paramos cronometro y medimos tiempo
    //un comentario mas porque puedo

    //Normalize the output values from 0.0 to 1.0
    cvScale(image2,image2,1.0/255.0); // re escalamos los valores
    //para que s epuedan imprimir o algo asi 

    cvShowImage("Original Image //xq hay que ocmentar hasta ne las ventanas de display", sharmuta );// te la enseño 
    cvShowImage("Sobeled Image //osea sobeleada", image2);// la imagen!!! 
    
    hipEventRecord( fn2, 0 ); // no se que poner 
    hipEventSynchronize( fn2 );// asi que pongo lo que es. sincronizamos eventos 
    hipEventElapsedTime( &tt2, st2, fn2 );//paramos cronometro y medimos tiempo
    printf("tiempo calculos en ms: %f\t tiempo de total %f\n", tiempo,tt2);//cazar elefantes... o imprimir no se solo soy un comentario


    cvWaitKey(0);// 我々は、任意のキーを押して、ユーザを待ちます
    
    return 0; // que malo no devuelves nada 
}
