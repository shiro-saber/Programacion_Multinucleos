#include <iostream>
#include <opencv2/opencv.hpp>
#include <stdio.h>
#include "cudaKernel.cu"

using namespace cv;
using namespace std;

int main(int argc, char** argv)
{
    IplImage* image;
    IplImage* sharmuta;

    image = cvLoadImage("cancer.jpg", CV_LOAD_IMAGE_GRAYSCALE);
    sharmuta = cvLoadImage("cancer.jpg", CV_LOAD_IMAGE_COLOR);

    if(!image )
    {
        cout << "Could not open or find the image" << std::endl;
        return -1;
    }


    IplImage* image2 = cvCreateImage(cvGetSize(image),IPL_DEPTH_32F,image->nChannels);
    IplImage* image3 = cvCreateImage(cvGetSize(image),IPL_DEPTH_32F,image->nChannels);

    //Convert the input image to float
    cvConvert(image,image3);

    float *output = (float*)image2->imageData;
    float *input =  (float*)image3->imageData;

    kernelcall(input, output, image->width,image->height, image3->widthStep);

    //Normalize the output values from 0.0 to 1.0
    cvScale(image2,image2,1.0/255.0);

    cvShowImage("Original Image", sharmuta );
    cvShowImage("Sobeled Image", image2);
    cvWaitKey(0);
    // namedWindow("Input",WINDOW_NORMAL);
    // namedWindow("Output",WINDOW_NORMAL);
    // imshow("Input",image);
    // imshow("Output",image2);
    // waitKey();
    return 0;
}
