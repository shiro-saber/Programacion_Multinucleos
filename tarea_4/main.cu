#include <iostream>
#include <opencv2/opencv.hpp>
#include <stdio.h>
#include "cudaKernel.cu"

using namespace cv;
using namespace std;

int main(int argc, char** argv)
{
    hipEvent_t inicio, fin,st2,fn2;
    float tiempo, tt2;
    
    hipEventCreate( &st2 );
    hipEventCreate( &fn2 );
    hipEventRecord( st2, 0 );

    IplImage* image;
    IplImage* sharmuta;

    image = cvLoadImage("salon.jpeg", CV_LOAD_IMAGE_GRAYSCALE);
    sharmuta = cvLoadImage("salon.jpeg", CV_LOAD_IMAGE_COLOR);

    if(!image )
    {
        cout << "Could not open or find the image" << std::endl;
        return -1;
    }


    IplImage* image2 = cvCreateImage(cvGetSize(image),IPL_DEPTH_32F,image->nChannels);
    IplImage* image3 = cvCreateImage(cvGetSize(image),IPL_DEPTH_32F,image->nChannels);

    //Convert the input image to float
    cvConvert(image,image3);

    float *output = (float*)image2->imageData;
    float *input =  (float*)image3->imageData;
    
    hipEventCreate( &inicio );
    hipEventCreate( &fin );
    hipEventRecord( inicio, 0 );//para medir el tiempo de EJECUCION

    kernelcall(input, output, image->width,image->height, image3->widthStep);
    
    hipEventRecord( fin, 0 );
    hipEventSynchronize( fin );
    hipEventElapsedTime( &tiempo, inicio, fin );//paramos cronometro y medimos tiempo

    //Normalize the output values from 0.0 to 1.0
    cvScale(image2,image2,1.0/255.0);

    cvShowImage("Original Image", sharmuta );
    cvShowImage("Sobeled Image", image2);
    
    hipEventRecord( fn2, 0 );
    hipEventSynchronize( fn2 );
    hipEventElapsedTime( &tt2, st2, fn2 );//paramos cronometro y medimos tiempo
    printf("tiempo calculos en ms: %f\t tiempo de total %f\n", tiempo,tt2);//cazar elefantes... o imprimir no se solo soy un comentario

    return 0;
}


    cvWaitKey(0);
    
    return 0;
}
