#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h> // un comentario se balanceaba sobre la tarea de moy
#include <iostream> // como veia que resisita fue a llamar otro comentario 

using namespace std; // 2 comentarios se balanceaban sobre la tarea de moy

#define CudaSafeCall( err ) __cudaSafeCall( err, __FILE__, __LINE__ ) // como veian que resisita fueron a llamar otro comentario // funciones para el manejo de errores con cuda
#define CudaCheckError()    __cudaCheckError( __FILE__, __LINE__ ) // 3 comentarios se balanceaban sobre la tarea de moy // funciones para el manejo de errores con cuda
#define checkCudaErrors(err) __checkCudaErrors (err, __FILE__, __LINE__) // como veian que resisita fueron a llamar otro comentario // funciones para el manejo de errores con cuda


texture <float,2,hipReadModeElementType> tex1; // 3 comentarios se balanceaban sobre la tarea de moy // declaramos al textura 

static hipArray *cuArray = NULL; // como veian que resisita fueron a llamar otro comentario // y claro que sea estatico para que las modificaciones y eso, ya sabes

//Kernel for x direction sobel
__global__ void implement_x_sobel(float* output,int width,int height,int widthStep) // 4 comentarios se balanceaban sobre la tarea de moy // osea sobel en x
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;// como veian que resisita fueron a llamar otro comentario // el index de x basicamente
    int y = blockIdx.y * blockDim.y + threadIdx.y;// 5 comentarios se balanceaban sobre la tarea de moy // lo mismo de y

    //Make sure that thread is inside image bounds
    if(x<width && y<height) // como veian que resisita fueron a llamar otro comentario // revisamos que el thread se ejecute dentro de la imagen
    {
        float output_value = (-1*tex2D(tex1,x-1,y-1)) + (0*tex2D(tex1,x,y-1)) + (1*tex2D(tex1,x+1,y-1)) // 6 comentarios se balanceaban sobre la tarea de moy
                           + (-2*tex2D(tex1,x-1,y))   + (0*tex2D(tex1,x,y))   + (2*tex2D(tex1,x+1,y)) // como veian que resisita fueron a llamar otro comentario
                           + (-1*tex2D(tex1,x-1,y+1)) + (0*tex2D(tex1,x,y+1)) + (1*tex2D(tex1,x+1,y+1)); // 7 comentarios se balanceaban sobre la tarea de moy

        output[y*widthStep+x]=output_value; // como veian que resisita fueron a llamar otro comentario 
    }

}


inline void __checkCudaErrors( hipError_t err, const char *file, const int line ) // 8 comentarios se balanceaban sobre la tarea de moy // errores chequeo 
{
    if( hipSuccess != err) {  // como veian que resisita fueron a llamar otro comentario
        fprintf(stderr, "%s(%i) : CUDA Runtime API error %d: %s.\n", // 9 comentarios se balanceaban sobre la tarea de moy
            file, line, (int)err, hipGetErrorString( err ) );  // como veian que resisita fueron a llamar otro comentario
        exit(-1);  // 10 comentarios se balanceaban sobre la tarea de moy
    }
}
// como veian que resisita fueron a llamar otro comentario
//Host Code
inline void __cudaSafeCall( hipError_t err, const char *file, const int line ) // 11 comentarios se balanceaban sobre la tarea de moy
{
#ifdef CUDA_ERROR_CHECK 
// como veian que resisita fueron a llamar otro comentario
    if ( hipSuccess != err ) // 12 comentarios se balanceaban sobre la tarea de moy
    {
        printf("cudaSafeCall() failed at %s:%i : %s\n", // como veian que resisita fueron a llamar otro comentario
            file, line, hipGetErrorString( err ) ); // 13 comentarios se balanceaban sobre la tarea de moy
        exit( -1 ); // como veian que resisita fueron a llamar otro comentario
    }
#endif  

    return;// 14 comentarios se balanceaban sobre la tarea de moy
}
inline void __cudaCheckError( const char *file, const int line ) // como veian que resisita fueron a llamar otro comentario
{
#ifdef CUDA_ERROR_CHECK
    // 15 comentarios se balanceaban sobre la tarea de moy
    hipError_t err = hipGetLastError();// como veian que resisita fueron a llamar otro comentario
    if ( hipSuccess != err ) // 16 comentarios se balanceaban sobre la tarea de moy
    {
        printf("cudaCheckError() failed at %s:%i : %s\n",// como veian que resisita fueron a llamar otro comentario
            file, line, hipGetErrorString( err ) );// 17 comentarios se balanceaban sobre la tarea de moy
        exit( -1 );// como veian que resisita fueron a llamar otro comentario
    }
#endif

    return;// 18 comentarios se balanceaban sobre la tarea de moy
}

void kernelcall(float* input,float* output,int width,int height,int widthStep, int blok, int threads)  // como veian que resisita fueron a llamar otro comentario // funcion prijcipal donde se llama al kernel de cuda, los parametros son las imagenes y b & t
{
    hipChannelFormatDesc channelDesc = hipCreateChannelDesc<float>(); // 19 comentarios se balanceaban sobre la tarea de moy // segun esto un canal de cuda, falta analizar esto en clase

    CudaSafeCall(hipMallocArray(&cuArray,&channelDesc,width,height)); // como veian que resisita fueron a llamar otro comentario // el tipico cuda malloc 

    //Never use 1D memory copy if host and device pointers have different widthStep.
    // You don't know the width step of CUDA array, so its better to use hipMemcpy2D...
    hipMemcpy2DToArray(cuArray,0,0,input,widthStep,width * sizeof(float),height,hipMemcpyHostToDevice); // 20 comentarios se balanceaban sobre la tarea de moy // copiamos la matriz 

    hipBindTextureToArray(tex1,cuArray,channelDesc);// 21 comentarios se balanceaban sobre la tarea de moy // le asignamos la textura que habiamos creado

    float * D_output_x; // como veian que resisita fueron a llamar otro comentario
    CudaSafeCall(hipMalloc(&D_output_x,widthStep*height)); // 22 comentarios se balanceaban sobre la tarea de moy
    /*
    dim3 blocksize(16,16); // 23 comentarios se balanceaban sobre la tarea de moy
    dim3 gridsize; // como veian que resisita fueron a llamar otro comentario
    gridsize.x=(width+blocksize.x-1)/blocksize.x; // 24 comentarios se balanceaban sobre la tarea de moy
    gridsize.y=(height+blocksize.y-1)/blocksize.y; // como veian que resisita fueron a llamar otro comentario
    */ // 25 comentarios se balanceaban sobre la tarea de moy // xq fuck it and fuck CUDA, vamos a usar nuestros propios bloques y threads 
    //implement_x_sobel<<<gridsize,blocksize>>>(D_output_x,width,height,widthStep/sizeof(float)); // como veian que resisita fueron a llamar otro comentario // la buena
    implement_x_sobel<<<blok,threads>>>(D_output_x,width,height,widthStep/sizeof(float)); // 26 comentarios se balanceaban sobre la tarea de moy // llamos la funcion del kernel, con los paramtros que debiaste ahber pasado desde la terminal, si los olvidaste vuelve a ver la terminal, ahi te lso recordamos

    hipDeviceSynchronize(); // como veian que resisita fueron a llamar otro comentario //sincornizamos despues del kernel 
    CudaCheckError(); // 27 comentarios se balanceaban sobre la tarea de moy // aseguramos que no haya avido errores 

    //Don't forget to unbind the texture
    hipUnbindTexture(tex1); // como veian que resisita fueron a llamar otro comentario // des bindeamos la textura e inventamos nuevas palabras 

    CudaSafeCall(hipMemcpy(output,D_output_x,height*widthStep,hipMemcpyDeviceToHost)); // 28 comentarios se balanceaban sobre la tarea de moy // "regresmos" la memoria al host

    hipFree(D_output_x); // como veian que resisita fueron a llamar otro comentario
    hipFreeArray(cuArray);// 29 comentarios se balanceaban sobre la tarea de moy
}
//como ya no resistio el codigo no llamaron a mas comentarios 