#include "hip/hip_runtime.h"
#include<hip/hip_runtime.h>
#include<iostream>
//#include "CudaKernel.h"

using namespace std;

#define CudaSafeCall( err ) __cudaSafeCall( err, __FILE__, __LINE__ )
#define CudaCheckError()    __cudaCheckError( __FILE__, __LINE__ )
#define checkCudaErrors(err) __checkCudaErrors (err, __FILE__, __LINE__)


texture <float,2,hipReadModeElementType> tex1;

static hipArray *cuArray = NULL;

//Kernel for x direction sobel
__global__ void implement_x_sobel(float* output,int width,int height,int widthStep)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    //Make sure that thread is inside image bounds
    if(x<width && y<height)
    {
        float output_value = (-1*tex2D(tex1,x-1,y-1)) + (0*tex2D(tex1,x,y-1)) + (1*tex2D(tex1,x+1,y-1))
                           + (-2*tex2D(tex1,x-1,y))   + (0*tex2D(tex1,x,y))   + (2*tex2D(tex1,x+1,y))
                           + (-1*tex2D(tex1,x-1,y+1)) + (0*tex2D(tex1,x,y+1)) + (1*tex2D(tex1,x+1,y+1));

        output[y*widthStep+x]=output_value;
    }

}


inline void __checkCudaErrors( hipError_t err, const char *file, const int line )
{
    if( hipSuccess != err) {
        fprintf(stderr, "%s(%i) : CUDA Runtime API error %d: %s.\n",
            file, line, (int)err, hipGetErrorString( err ) );
        exit(-1);
    }
}

//Host Code
inline void __cudaSafeCall( hipError_t err, const char *file, const int line )
{
#ifdef CUDA_ERROR_CHECK
    if ( hipSuccess != err )
    {
        printf("cudaSafeCall() failed at %s:%i : %s\n",
            file, line, hipGetErrorString( err ) );
        exit( -1 );
    }
#endif

    return;
}
inline void __cudaCheckError( const char *file, const int line )
{
#ifdef CUDA_ERROR_CHECK
    hipError_t err = hipGetLastError();
    if ( hipSuccess != err )
    {
        printf("cudaCheckError() failed at %s:%i : %s\n",
            file, line, hipGetErrorString( err ) );
        exit( -1 );
    }
#endif

    return;
}

void kernelcall(float* input,float* output,int width,int height,int widthStep)
{
    hipChannelFormatDesc channelDesc = hipCreateChannelDesc<float>();

    CudaSafeCall(hipMallocArray(&cuArray,&channelDesc,width,height));

    //Never use 1D memory copy if host and device pointers have different widthStep.
    // You don't know the width step of CUDA array, so its better to use hipMemcpy2D...
    hipMemcpy2DToArray(cuArray,0,0,input,widthStep,width * sizeof(float),height,hipMemcpyHostToDevice);

    hipBindTextureToArray(tex1,cuArray,channelDesc);

    float * D_output_x;
    CudaSafeCall(hipMalloc(&D_output_x,widthStep*height));

    dim3 blocksize(16,16);
    dim3 gridsize;
    gridsize.x=(width+blocksize.x-1)/blocksize.x;
    gridsize.y=(height+blocksize.y-1)/blocksize.y;

    implement_x_sobel<<<gridsize,blocksize>>>(D_output_x,width,height,widthStep/sizeof(float));

    hipDeviceSynchronize();
    CudaCheckError();

    //Don't forget to unbind the texture
    hipUnbindTexture(tex1);

    CudaSafeCall(hipMemcpy(output,D_output_x,height*widthStep,hipMemcpyDeviceToHost));

    hipFree(D_output_x);
    hipFreeArray(cuArray);
}
