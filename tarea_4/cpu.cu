#include "hip/hip_runtime.h"
#include<iostream>
#include<cmath>
#include<opencv2/imgproc/imgproc.hpp>
#include<opencv2/highgui/highgui.hpp>
 
 using namespace std;
 using namespace cv;
  
   
// Computes the x component of the gradient vector
// at a given point in a image.
// returns gradient in the x direction
int xGradient(Mat image, int x, int y)
{
  return image.at<uchar>(y-1, x-1) + 
         2*image.at<uchar>(y, x-1) + 
         image.at<uchar>(y+1, x-1) - 
         image.at<uchar>(y-1, x+1) - 
         2*image.at<uchar>(y, x+1) - 
         image.at<uchar>(y+1, x+1);
}
    
// Computes the y component of the gradient vector
// at a given point in a image
// returns gradient in the y direction
int yGradient(Mat image, int x, int y)
{
  return image.at<uchar>(y-1, x-1) +
         2*image.at<uchar>(y-1, x) +
         image.at<uchar>(y-1, x+1) -
         image.at<uchar>(y+1, x-1) -
         2*image.at<uchar>(y+1, x) -
         image.at<uchar>(y+1, x+1);
}
      
int main()
{
  Mat src, src2, dst;
  int gx, gy, sum;
                      
  // Load an image
  src = imread("cancer.jpg", CV_LOAD_IMAGE_GRAYSCALE);
  src2 = imread("cancer.jpg", CV_LOAD_IMAGE_COLOR);
  dst = src.clone();
  
  if( !src.data ) return -1;
                                                                                                        
  for(int y = 0; y < src.rows; y++)
    for(int x = 0; x < src.cols; x++)
      dst.at<uchar>(y,x) = 0.0;
                                                                                                 
  for(int y = 1; y < src.rows - 1; y++){
    for(int x = 1; x < src.cols - 1; x++){
      gx = xGradient(src, x, y);
      gy = yGradient(src, x, y);
      sum = abs(gx) + abs(gy);
      sum = sum > 255 ? 255:sum;
      sum = sum < 0 ? 0 : sum;
      dst.at<uchar>(y,x) = sum;
    }
  }
                                                                                                                                                        
  namedWindow("final");
  imshow("final", dst);
                                                                                                                    
  namedWindow("initial");
  imshow("initial", src2);

  waitKey();
          
  return 0;
}
