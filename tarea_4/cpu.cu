#include "hip/hip_runtime.h"
#include<iostream>
#include"stdio.h"
#include<cmath>
#include<opencv2/imgproc/imgproc.hpp>
#include<opencv2/highgui/highgui.hpp>
 
using namespace std; //un comentario para algo que hacemos siempre
using namespace cv; //si, este es para incluir el namespace cv de opencv
  
/* Saca el gradiente de x, en el punto dado en la direccion de x*/
int xGradient(Mat image, int x, int y)
{
  return image.at<uchar>(y-1, x-1) + 
         2*image.at<uchar>(y, x-1) + 
         image.at<uchar>(y+1, x-1) - 
         image.at<uchar>(y-1, x+1) - 
         2*image.at<uchar>(y, x+1) - 
         image.at<uchar>(y+1, x+1); // return gigantezco de las operaciones que se ejecutan
}
    
/* lo mismo del de arriba pero ahora cambia las x por y */
int yGradient(Mat image, int x, int y)
{
  return image.at<uchar>(y-1, x-1) +
         2*image.at<uchar>(y-1, x) +
         image.at<uchar>(y-1, x+1) -
         image.at<uchar>(y+1, x-1) -
         2*image.at<uchar>(y+1, x) -
         image.at<uchar>(y+1, x+1);// mismo return gigantezco que se veia feo en una linea
}
      
int main() //¿qué será esto?
{
  Mat src, src2, dst; //pos las imágenes
  int gx, gy, sum; // dejare un comentario en esta declaracion
  hipEvent_t inicio, fin,st2,fn2; // cida events \o/
  float tiempo, tiempo2; // dios mio, ¿que será?
                      
  // Carguemos la imagen
  src = imread("salon.jpeg", CV_LOAD_IMAGE_GRAYSCALE); // es más fácil hacerlo asi
  src2 = imread("salon.jpeg", CV_LOAD_IMAGE_COLOR); // pos necesitas mostrarla en algún momento no?
  dst = src.clone(); //hagamos un clon
  
  hipEventCreate( &st2 ); //mummy
  hipEventCreate( &fn2 ); //dracula
  hipEventRecord( st2, 0 ); //frankie

  if( !src.data ) return -1; // no manches, pasame algo
                                                                                                        
  for(int y = 0; y < src.rows; y++) //recorramos las filas
    for(int x = 0; x < src.cols; x++) //recorramos las columnas
      dst.at<uchar>(y,x) = 0.0; //punto inicial
  
  hipEventCreate( &inicio );// esta vivo!!!
  hipEventCreate( &fin ); // el bebe esta vivo!!!
  hipEventRecord( inicio, 0 );//para medir el tiempo de EJECUCION

  for(int y = 1; y < src.rows - 1; y++){
    for(int x = 1; x < src.cols - 1; x++){
      gx = xGradient(src, x, y); //dame el gradiente x
      gy = yGradient(src, x, y); //dame el gradiente y
      sum = abs(gx) + abs(gy); //suma que pues tenía que hacer
      sum = sum > 255 ? 255:sum; //ahora si la suma es mayor a 255 cierralo en 255 si no pues dejalo en paz
      sum = sum < 0 ? 0 : sum; // y que tal si es menor a 0 pos no te vas a los negativos
      dst.at<uchar>(y,x) = sum; // y vamos pasando los puntos.
    }
  }
  hipEventRecord( fin, 0 ); // esta muerto ¬¬
  hipEventSynchronize( fin ); //sincronizadas
  hipEventElapsedTime( &tiempo, inicio, fin );//paramos cronometro y medimos tiempo

                                                                                                                                                        
  namedWindow("final"); // hagamos un muñeco, digo una ventana
  imshow("final", dst); // print a la imagen generada 
                                                                                                                    
  namedWindow("initial"); //ya me puedo ir? 
  imshow("initial", src2); //neta lo tengo que comentar otra vez?
  
  hipEventRecord( fn2, 0 ); //dont kill me, please!
  hipEventSynchronize( fn2 ); //free the nipple
  hipEventElapsedTime( &tiempo2, st2, fn2 );//paramos cronometro y medimos tiempo
  
  printf("tiempo calculos en ms: %f\t tiempo de total %f\n", tiempo,tiempo2);//cazar elefantes... o imprimir no se solo soy un comentario

  waitKey(); //no se si esto es demasiado obvio para comentarlo
          
  return 0; // y yo? que hago aqui?
}
