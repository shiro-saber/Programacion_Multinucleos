
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>
#include <math.h>

#define PI 3.14159265358979323846264338327 //pi

FILE *output; //guardaremos los datos

int main (int argc, char* argv[])
{
	int numx, numt;
	if(argc < 3)
	{
		numx=500;  //# de puntos del grid
		numt=2000; //# de lapsos de tiempo
  }
	else
	{
  	numx = atoi(argv[1]);	 //puntos de la malla
  	numt = atoi(argv[1]);	//lapsos de tiempo
  }

	double dx=1/(numx-1);
	double dt=0.00005;
	double C[numx][numt];
	double x=0.0;
	double t;
	int i, j;
	hipEvent_t inicio, fin,st2,fn2; // cida events \o/
  float tiempo, tiempo2; // dios mio, ¿que será?
	double mu=0.5; //Parametros de GAUSS
	double sigma=0.05; //Distribucion inicial

	hipEventCreate( &st2 ); //mummy
  hipEventCreate( &fn2 ); //dracula
  hipEventRecord( st2, 0 ); //frankie

	output=fopen("d2d.dat", "w"); //output file

	C[0][0]=0.0; //el inicio siempre es 0
	C[numx-1][0]=0.0; //el final igual
	dx=1.0/(numx-1.0); //será una derivada?

	hipEventCreate( &inicio );// esta vivo!!!
  hipEventCreate( &fin ); // el bebe esta vivo!!!
  hipEventRecord( inicio, 0 );//para medir el tiempo de EJECUCION

	for(i=0; i<numx; i++){
	  x=i*dx;
	  C[i][0]=exp(-pow((x-mu),2.0)/(2.0*pow(sigma,2.0)))/pow((2.0*PI*pow(sigma,2.0)),0.5);  //condicion inicial para C=C(x,0)=gauss
	  C[0][0]=0.0; //condicion de frontera i)
	  C[numx-1][0]=0.0; //condicion de frontera ii)
	}

	for(j=0;j<numt;j++){ //main time stepping loop
	   t+=dt;
	  for(i=1; i<numx-1; i++){
	     x=i*dx;
	     C[i][j+1] = C[i][j] + (dt/pow(dx,2))*(C[i+1][j] - 2*C[i][j] + C[i-1][j]);
	  }
	 	C[0][j]=0.0; //condicion de frontera i)
	  C[numx-1][j]=0.0; //condicion de frontera ii)
	}

	hipEventRecord( fin, 0 ); // esta muerto ¬¬
  hipEventSynchronize( fin ); //sincronizadas
  hipEventElapsedTime( &tiempo, inicio, fin );//paramos cronometro y medimos tiempo

	C[10][0]=0.0; //stg wrong with inital condition from this point on******

	for(i=0; i<numx; i++){  // escribimos los datos del arreglo en el archivo
		x=i*dx;
	  fprintf(output, "%e\t", x);
	  for(j=0; j<numt; j++){
	  	fprintf(output, "%e\t", C[i][j]);
	  }
		fprintf(output, "\n");
	}

	fflush(output);
	fclose(output); //close output file

	hipEventRecord( fn2, 0 ); //dont kill me, please!
  hipEventSynchronize( fn2 ); //free the nipple
  hipEventElapsedTime( &tiempo2, st2, fn2 );//paramos cronometro y medimos tiempo

  printf("tiempo calculos en ms: %f\t tiempo de total %f\n", tiempo,tiempo2);//cazar elefantes... o imprimir no se solo soy un comentario

	return 0;
}
