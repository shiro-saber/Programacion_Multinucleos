#include "hip/hip_runtime.h"
//////////////////////////////////////////////////////////////////
//Templete 2D para la ecuacion de calor sin uso de memoria de textura
//Basado en el programa demo:
//http://www.many-core.group.cam.ac.uk/projects/LBdemo.shtml
//Autor: Carlos Malaga
//Para compilar en CUDA 2.3
//nvcc -o templete Templete2D.cu -I$HOME/NVIDIA_GPU_Computing_SDK/C/common/inc 
//////////////////////////////////////////////////////////////

// Librerias
#include <stdio.h>
#include <stdlib.h>
#include <cutil.h>

// Tamaño del bloque
#define TILE_I 16
#define TILE_J 16

// Punteros en el CPU 
float *t0; 

// Punteros, arreglos y texturas en el GPU    
float             *t_data;
float         *t_data_old; 


// Escalares globales
float dx,dy,dp,dt,kcond;
int   ni,nj,paso;
size_t pitch;

// Definicion del CUDA kernel 
__global__ void solveheat_kernel (int ni, int nj, int pitch, float kcond, float dt, float dx, float dy, float *t_data, float *t_data_old);

// Definicion C wrappers
void solveheat(void);
void Imprimir(void);

///////////////////////////////////////////////////////////////////

int main(void)
{
    int i;
    int totpoints;


    dt = 0.01f;
    dx = 0.1f;
    dy = 0.1f;
    kcond = 0.01f;
    ni=800;
    nj=800;
    totpoints = ni*nj;

    printf ("ni = %d\n", ni);
    printf ("nj = %d\n", nj);
    printf ("Numero de puntos = %d\n", totpoints);
    
    // Asigna la memoria en el CPU (host)
    t0 = (float *)malloc(ni*nj*sizeof(float));

    // Asigna la memoria en el GPU (device)
    hipMallocPitch((void **)&t_data, &pitch, sizeof(float)*ni, nj);
    hipMallocPitch((void **)&t_data_old, &pitch, sizeof(float)*ni, nj);


    // Valores iniciales del campo t
    for (i=0; i<totpoints; i++) {
	t0[i] = 0.f;
    }
    t0[totpoints/2 + ni/2] = 1000.f;

    // Copia valores iniciales al GPU
    hipMemcpy2D((void *)t_data, pitch, (void *)t0,sizeof(float)*ni,sizeof(float)*ni, nj,
                                hipMemcpyHostToDevice);
   
    paso = 0;
    
    for (i=1;i<=10000;i++){    
    paso = paso + 1;   
    solveheat();   
    if (paso%1000 == 0) printf ("Iteracion: %d\n", paso);  
    }
    	
    Imprimir();

    // 	
    printf("CUDA: %s\n", hipGetErrorString(hipGetLastError()));
    
    return 0;
}

////////////////////////////////////////////////////////////////////////////////

void solveheat(void)
{
    // Copiado de t_data a t_array y "Bind" de t_array a la textura	
   
  hipMemcpy2D((void *)t_data_old, pitch, (void *)t_data,sizeof(float)*ni,sizeof(float)*ni, nj,
                                hipMemcpyDeviceToDevice);
   
	

    dim3 grid = dim3(ni/TILE_I, nj/TILE_J);
    dim3 block = dim3(TILE_I, TILE_J);

    solveheat_kernel<<<grid, block>>>(ni,nj,pitch,kcond,dt,dx,dy,t_data, t_data_old);


}

__global__ void solveheat_kernel (int ni,int nj,int pitch, float kcond, float dt, 
                                                float dx, float dy, float *t_data, float *t_data_old)
{
    int i, j, i2d, i2d2, i2d3, i2d4, i2d5;
    float told,tnow,tip1,tim1,tjp1,tjm1;
    
    i = blockIdx.x*TILE_I + threadIdx.x;
    j = blockIdx.y*TILE_J + threadIdx.y;

    i2d = i + j*pitch/sizeof(float);
    i2d2= (i+1) + (j)*pitch/sizeof(float);
    i2d3= (i-1) + (j)*pitch/sizeof(float);
    i2d4= (i) + (j+1)*pitch/sizeof(float);
    i2d5= (i) + (j-1)*pitch/sizeof(float);
    
    if (i ==ni-1) i2d2= ni-1 + (j)*pitch/sizeof(float);
    if (i == 0) i2d3= 0 + (j)*pitch/sizeof(float);
    if (j ==nj-1) i2d4= i + (nj-1)*pitch/sizeof(float);
    if (j == 0) i2d5= i + (0)*pitch/sizeof(float);

    told = t_data_old[i2d];
    tip1 = t_data_old[i2d2];
    tim1 = t_data_old[i2d3];
    tjp1 = t_data_old[i2d4];
    tjm1 = t_data_old[i2d5];

    tnow = told + dt*kcond*((tip1-2.0f*told+tim1)/(dx*dx) 
                          + (tjp1-2.0f*told+tjm1)/(dy*dy));
    t_data[i2d] = tnow;
}

////////////////////////////////////////////////////////////////////////////////

void Imprimir(void)
{
      
    int i, j, i2d;
    float t; 
    FILE *fp;

    // Copia de VRAM a RAM
    hipMemcpy((void *)t0, (void *)t_data, nj*ni*sizeof(float), hipMemcpyDeviceToHost);

    fp = fopen ( "Datos_sintex.dat", "w+" );
    
    for (i=0;i<ni;++i){
      for (j=0;j<nj;++j){
      i2d = i + ni*j;
      t = t0[i2d];
	fprintf(fp, "%f\t %f\t %f\n" , i*dx,j*dy,t);
      }
      fprintf(fp, "\n");
    }
    fclose ( fp );
}    

////////////////////////////////////////////////////////////////////////////////


