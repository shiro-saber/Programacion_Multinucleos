#include "hip/hip_runtime.h"
#include <stdio.h>
#include "book.h"

/* experiment with N */
/* how large can it be? */
#define N 1000000
#define THREADS_PER_BLOCK 1000

__global__ void add(int *a, int *b, int *c, int size)
{
   int index = blockIdx.x * blockDim.x + threadIdx.x;
   if (index < size)
      c[index] = a[index] + b[index];
}

struct DataStruct {
    int    deviceID;
    int    size;
    int   *a;
    int   *b;
    int   *returnC;
};

void * sumaGPUS(void* dataI)
{
  DataStruct  *data = (DataStruct*)dataI;
  HANDLE_ERROR( hipSetDevice( data->deviceID ) ); // 0 o 1

   int *a, *b, *c;
   int *d_a, *d_b, *d_c;
   int size = N * sizeof( int );
   float tiempo1, tiempo2;
   hipEvent_t inicio1, fin1, inicio2, fin2; // para medir tiempos como con timestamp

   /* allocate space for host copies of a, b, c and setup input alues */

   a = (int *)malloc( size );
   b = (int *)malloc( size );
   c = (int *)malloc( size );

   for( int i = 0; i < N; i++ )
      a[i] = b[i] = i+1;

   /*
   hipEventCreate(&inicio1); // Se inicializan
   hipEventCreate(&fin1);
   hipEventRecord( inicio1, 0 ); // Se toma el tiempo de inicio
   */

   /* allocate space for device copies of a, b, c */

   hipMalloc( (void **) &d_a, size );
   hipMalloc( (void **) &d_b, size );
   hipMalloc( (void **) &d_c, size );

   /* copy inputs to deice */
   /* fix the parameters needed to copy data to the device */
   hipMemcpy( d_a, a, size, hipMemcpyHostToDevice );
   hipMemcpy( d_b, b, size, hipMemcpyHostToDevice );

   /*
   hipEventCreate(&inicio2); // Se inicializan
   hipEventCreate(&fin2);
   hipEventRecord( inicio2, 0 ); // Se toma el tiempo de inicio
   */

   /* launch the kernel on the GPU */
   add<<< N / THREADS_PER_BLOCK, THREADS_PER_BLOCK >>>( d_a, d_b, d_c, data->size );

   /*
   hipEventRecord( fin2, 0); // Se toma el tiempo final.
   hipEventSynchronize( fin2 ); // Se sincroniza
   hipEventElapsedTime( &tiempo2, inicio2, fin2 );
   */

   /* copy result back to host */
   /* fix the parameters needed to copy data back to the host */
   hipMemcpy( c, d_c, size, hipMemcpyDeviceToHost );

   hipFree( d_a );
   hipFree( d_b );
   hipFree( d_c );

   /*
   hipEventRecord( fin1, 0); // Se toma el tiempo final.
   hipEventSynchronize( fin1 ); // Se sincroniza
   hipEventElapsedTime( &tiempo1, inicio1, fin1 );
   */

   //for (int i=0; i<N; i++)
  //    printf( "%d + %d = %d\n", a[i], b[i], c[i] );

   /* clean up */

   free(a);
   free(b);
   free(c);

   /*
   printf("Tiempo cálculo %f ms\n", tiempo2);
   printf("Tiempo total %f ms\n", tiempo1);
   */


} /* end main */

int main(void){
  float tiempo1, tiempo2;
  hipEvent_t inicio1, fin1, inicio2, fin2; // para medir tiempos como con timestamp

hipDeviceProp_t prop;
int numVideocards=0;
hipGetDeviceCount(&numVideocards);
hipGetDeviceProperties(&prop,1);
printf("num cards: %d \n",numVideocards);

int   *a = (int*)malloc( sizeof(int) * N );
HANDLE_NULL( a );
int   *b = (int*)malloc( sizeof(int) * N );
HANDLE_NULL( b );
int *c  = (int*)malloc( sizeof(int) * N );
int * kuz = (int*)malloc( sizeof(int) * N );
HANDLE_NULL( c );
// fill in the host memory with data
for (int i=0; i<N; i++) {
    a[i] = i;
    b[i] = i*2;
}

hipEventCreate(&inicio1); // Se inicializan
hipEventCreate(&fin1);
hipEventRecord( inicio1, 0 ); // Se toma el tiempo de inicio

// prepare for multithread
DataStruct  data[2];
data[0].deviceID = 0;
data[0].size = N*3/4;
data[0].a = a;
data[0].b = b;
data[0].returnC = c;

data[1].deviceID = 1;
data[1].size = N/4;
data[1].a = a + N/2;
data[1].b = b + N/2;
data[1].returnC = c + N/2;

CUTThread   thread = start_thread( sumaGPUS, &(data[0]) );
sumaGPUS( &(data[1]) );
end_thread( thread );

hipEventRecord( fin1, 0); // Se toma el tiempo final.
hipEventSynchronize( fin1 ); // Se sincroniza
hipEventElapsedTime( &tiempo1, inicio1, fin1 );
kuz[0]=*data[0].returnC;
kuz[N*3/4] =* data[1].returnC;
//for (int i=0; i<N; i++)
//    printf( "%d + %d = %d\n", a[i], b[i], c[i] );
for (int i=0; i<N; i++)
    printf( "%d + %d = %d\n", a[i], b[i], kuz[i] );

//printf("Tiempo cálculo %f ms\n", tiempo2);
printf("Tiempo allah %f ms\n", tiempo1);

  return 0;
}
