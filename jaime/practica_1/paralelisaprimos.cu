#include "hip/hip_runtime.h"
#include <stdio.h>
#include <math.h>
#include <stdlib.h>


#define THREADS 1000
#define MAX 100000000

__global__ void primos (char * d_arr2, int *d_raiz){
	
	//int index = blockIdx.x * blockDim.x + threadIdx.x;
	//int raiz = sqrt(MAX);
	__shared__ char *d_arr;
	__syncthreads();
	d_arr = d_arr2;
	//__shared__ float cache[threadsPerBlock];
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    int cacheIndex = threadIdx.x;

    if(d_arr2[tid] ==0)
	    while(tid*tid < MAX){

	    	d_arr2[tid*tid] =1;
	    	tid += blockDim.x * gridDim.x;
	    }

	
__syncthreads();
	printf("Run kernel: %s\n", hipGetErrorString(hipDeviceSynchronize()));
}


int main(int argc, char *argv[])
{
	hipEvent_t st1, fn1, st2,fn2;
	float tt1, tt2;

	hipEventCreate( &st1 );
	hipEventCreate( &fn1 );
	hipEventRecord( st1, 0 );

	unsigned long i, j, c=0;
	char *arr = new char[MAX];
	char *d_arr;
	int *d_raiz;
	hipMalloc((void**) &d_arr, MAX* sizeof(char));
	hipMalloc((void**) &d_raiz, sizeof(int));
	int *raiz;
	*raiz = sqrt(MAX);


	for (i=0; i<MAX; i++)
		arr[i] = 0;
	arr[0] = 1;
	arr[1] = 1;

	hipMemcpy(d_arr,arr,MAX*sizeof(char), hipMemcpyHostToDevice);
	hipMemcpy(d_raiz,raiz,sizeof(int), hipMemcpyHostToDevice);


	hipEventCreate( &st2 );
	hipEventCreate( &fn2 );
	hipEventRecord( st2, 0 );

	primos<<<MAX/THREADS,THREADS>>>(d_arr,d_raiz);

	hipEventRecord( fn2, 0 );
      hipEventSynchronize( fn2 );
      hipEventElapsedTime( &tt2, st2, fn2 );

hipMemcpy(arr,d_arr, MAX*sizeof(char), hipMemcpyDeviceToHost);
int cont =0;

      for (int i = 0; i < MAX; ++i)
      {
      	if(d_arr[i]==0){
      		cont++;
      	}

      }
printf("%ld\n", cont);

hipFree(d_arr);
hipFree(d_raiz);
free(arr);


	return 1;
}