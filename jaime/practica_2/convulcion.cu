
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

#define M 32

int t1,t2,t3;

void desplegar(int *m);
//funcion para la multiplicacion de matrices, el kernel
__global__ void calcularGPU3D(int *m1, int *m2, int *m3, int n)
{
   int i = blockIdx.x*blockDim.x + threadIdx.x;
   int j = blockIdx.y*blockDim.y + threadIdx.y;
   int k = blockIdx.z*blockDim.z + threadIdx.z;
   m3[i*n+j] = 0;
   __syncthreads();
   m3[i*n+j] += m1[i*n+k] * m2[k*n+j];
}

short n;
hipEvent_t gpuI, gpuF;
float gpuT;
//llamada al kernel desde cpu
float multiplicarGPU(int *a, int *b, int *c)
{
   int *dev_a, *dev_b, *dev_c;
   hipError_t err;

   hipEventCreate( &gpuI );

   hipEventCreate( &gpuF );
   hipEventRecord( gpuI, 0 );
//funciones de mem y mallooc con sus errores cachados
   err=hipMalloc( (void**)&dev_a, n*n*sizeof(int) );
   printf("CUDA malloc 1: %s\n",hipGetErrorString(err));
   err=hipMalloc( (void**)&dev_b, n*n*sizeof(int) );
   printf("CUDA malloc 2: %s\n",hipGetErrorString(err));
   err=hipMalloc( (void**)&dev_c, n*n*sizeof(int) );
   printf("CUDA malloc 3: %s\n",hipGetErrorString(err));
   err=hipMemcpy( dev_a, a, n*n*sizeof(int), hipMemcpyHostToDevice );
   printf("CUDA mem copy: %s\n",hipGetErrorString(err));
   err=hipMemcpy( dev_b, b, n*n*sizeof(int), hipMemcpyHostToDevice );
   printf("CUDA mem copy: %s\n",hipGetErrorString(err));


      dim3 bloques( t2/M, t2/M, t2/M );
      dim3 threads( M, M, M );
      calcularGPU3D<<<bloques, threads>>>( dev_a, dev_b, dev_c, n );

   err=hipDeviceSynchronize();
   printf("CUDA sync: %s\n",hipGetErrorString(err));
   err=hipMemcpy( c, dev_c, n*n*sizeof(int), hipMemcpyDeviceToHost );
   printf("CUDA copy a CPU: %s\n",hipGetErrorString(err));

   hipEventRecord( gpuF, 0 );
   hipEventSynchronize( gpuF );
   hipEventElapsedTime( &gpuT, gpuI, gpuF );
   hipFree( dev_a );
   hipFree( dev_b );
   hipFree( dev_c );
   return gpuT;
}

void desplegar(int *m)
{
   for (int i=0; i<n; i++)
   {
      for (int j=0; j<n; j++)
         printf("%d ", m[i*n+j]);
      printf("\n");
   }
   printf("\n");
}// buscando elefantes para cazarlos

//rellenamos las matrices
void inicializar(int *a, int *b, int *c)
{
   for(int i = 0; i<t2*t3; i++)
   {
      c[i] = 0;
   }//llenamos c con puro 0
   bool suma=true;
   int kuz=1;
   //llenamos b con filas de 1 al 255 ida y vvuelta ahsta que quede completo
   for(int i =0; i < t2*t3; i++){
     if (suma){
       b[i] = kuz;
       kuz++;
     }else{
       b[i]=kuz;
       kuz--;
     }
     if(i%255 == 0){
       suma = !suma;
     }
   }
   // llenar la matriz pequeña con 1 2 y 3
   for (int i=0;i<t1;i++){
   a[i]= 1;
   a[i+1]=2;
   a[i+2]=3;
 }
}


//llamar a la funcion que hace los mallocs y llama al kernel de cuda, algo asi como un proxy
void multiplicarMatrices( int *a, int *b, int *c)
{

   //int *a, *b, *c;



      multiplicarGPU( a, b, c) ;

    //desplegar(c);
   free( a );
   free( b );
   free( c );
}


//obtener la mini matriz de la mega matriz para poder ahcer la multiplicacion de matrizita con matriz pequeña
//osea, sacarle el pedazo a la matriz para sobreponerlo con  la otra
void minime(int *x,int *r,int iter){
  for(int i =0; i < t1;i++){
    for(int j=0; i < t1;j++){
      r[i+j] = x[(iter*i*j)];
    }
  }
}

//saca la suma y divicion de los elementos de la matriz resultante
int sumamela(int *d){
  int zain=0;
  for(int i =0; i< t1*t1; i++){
    zain += d[i];
  }
  zain /=(t1*t1);
  return zain;
}

// algo asi como el main, pero como no te gustan mains largos aki ta
void convulcion(){
  int *a, *b, *c, *d;//d es la matriz reducida de b para "sobnreponer" con a
  a = (int*) malloc(t1*sizeof(int));
  b = (int*) malloc(t2*t3*sizeof(int));
  c = (int*) malloc(t2*t3*sizeof(int));
//printf("inici\n");
  int iter =1;
  inicializar( a, b, c );
  //printf("inicializado\n");
  while(iter < (t2-1)*(t3-1))
  {
    //printf("problema\n");
  minime(b,d, iter);

  multiplicarMatrices(a,d,c);
  iter++;
  b[t2+1+iter] = sumamela(d);
  }

}

hipEvent_t ts,tf ;
float tt;

int main (int argc, char *argv[] )
{
  hipEventCreate( &ts );
  hipEventCreate( &tf );
  hipEventRecord( ts, 0 );
//iniciamos timers
//cachamos args del comando
   if ( argv[1] == NULL || argv[2]== NULL || argv[3]==NULL )
   {
      printf("insuficientes aprametros \n");
      return -3;
   }

   if(atoi(argv[1])%2 ==0 || atoi(argv[2])< 512 || atoi(argv[3])< 256)
    return -3;

    t1 = atoi(argv[1]);
    t2 = atoi(argv[2]);
    t3 = atoi(argv[3]);
   //multiplicarMatrices( atoi(argv[1]), atoi(argv[2]), atoi(argv[3]) );
   convulcion(); // nos vamos de cazeria yey
   hipEventRecord( tf, 0 );
   hipEventSynchronize( tf );
   hipEventElapsedTime( &tt, ts, tf );
   printf("tiempo de calculo: %f \t tiempo total: %f \n", gpuT,tt); /// JAJAJAJAJ jamas llegaras a esta linea porque de PI#$%&# segmentation fault
   return 1;
}
