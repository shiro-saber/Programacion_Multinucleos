#include <iostream> 
#include "thrust_samples.h"

int main(void)
{
	int major = THRUST_MAJOR_VERSION;
	int minor = THRUST_MINOR_VERSION;

	std::cout << "Thrust v" << major << "." << minor << std::endl;
	
	/*generateThrustSequence();
	Thrust_sort();
	Thrust_basic_transformations();
	Thrust_function_transform();
	Thrust_zip_iterator();*/
	Thrust_zip_pointers();
	std::cin.get();
}
