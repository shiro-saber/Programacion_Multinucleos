
#include <hip/hip_runtime.h>
#include <stdio.h>

/* experiment with N */
/* how large can it be? */
#define N 1000000
#define THREADS_PER_BLOCK 1000

__global__ void add(int *a, int *b, int *c)
{
   int index = blockIdx.x * blockDim.x + threadIdx.x;
   if (index < N)
      c[index] = a[index] + b[index];
}

int main()
{
   int *a, *b, *c;
   int *d_a, *d_b, *d_c;
   int size = N * sizeof( int );
   float tiempo1, tiempo2;
   hipEvent_t inicio1, fin1, inicio2, fin2; // para medir tiempos como con timestamp

   /* allocate space for host copies of a, b, c and setup input alues */

   a = (int *)malloc( size );
   b = (int *)malloc( size );
   c = (int *)malloc( size );

   for( int i = 0; i < N; i++ )
      a[i] = b[i] = i+1;

   hipEventCreate(&inicio1); // Se inicializan
   hipEventCreate(&fin1);
   hipEventRecord( inicio1, 0 ); // Se toma el tiempo de inicio

   /* allocate space for device copies of a, b, c */

   hipMalloc( (void **) &d_a, size );
   hipMalloc( (void **) &d_b, size );
   hipMalloc( (void **) &d_c, size );

   /* copy inputs to deice */
   /* fix the parameters needed to copy data to the device */
   hipMemcpy( d_a, a, size, hipMemcpyHostToDevice );
   hipMemcpy( d_b, b, size, hipMemcpyHostToDevice );

   hipEventCreate(&inicio2); // Se inicializan
   hipEventCreate(&fin2);
   hipEventRecord( inicio2, 0 ); // Se toma el tiempo de inicio

   /* launch the kernel on the GPU */
   add<<< N / THREADS_PER_BLOCK, THREADS_PER_BLOCK >>>( d_a, d_b, d_c );

   hipEventRecord( fin2, 0); // Se toma el tiempo final.
   hipEventSynchronize( fin2 ); // Se sincroniza
   hipEventElapsedTime( &tiempo2, inicio2, fin2 );

   /* copy result back to host */
   /* fix the parameters needed to copy data back to the host */
   hipMemcpy( c, d_c, size, hipMemcpyDeviceToHost );

   hipFree( d_a );
   hipFree( d_b );
   hipFree( d_c );

   hipEventRecord( fin1, 0); // Se toma el tiempo final.
   hipEventSynchronize( fin1 ); // Se sincroniza
   hipEventElapsedTime( &tiempo1, inicio1, fin1 );

   for (int i=0; i<N; i++)
      printf( "%d + %d = %d\n", a[i], b[i], c[i] );

   /* clean up */

   free(a);
   free(b);
   free(c);

   printf("Tiempo cálculo %f ms\n", tiempo2);
   printf("Tiempo total %f ms\n", tiempo1);
	
   return 0;
} /* end main */
