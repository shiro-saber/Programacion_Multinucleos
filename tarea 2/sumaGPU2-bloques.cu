/*
 * Copyright 1993-2010 NVIDIA Corporation.  All rights reserved.
 *
 * NVIDIA Corporation and its licensors retain all intellectual property and 
 * proprietary rights in and to this software and related documentation. 
 * Any use, reproduction, disclosure, or distribution of this software 
 * and related documentation without an express license agreement from
 * NVIDIA Corporation is strictly prohibited.
 *
 * Please refer to the applicable NVIDIA end user license agreement (EULA) 
 * associated with this source code for terms and conditions that govern 
 * your use of this NVIDIA software.
 * 
 */



#include <hip/hip_runtime.h>
#include <stdio.h>

#define N   1000
#define M   1000000
int d;

__global__ void add( int *a, int *b, int *c, int i ) {
   int tid = i*N+blockIdx.x; // vector index
   if (tid < M)
      c[tid] = a[tid] + b[tid];
}

int main( void ) {
   int *a= new int[M], *b=new int[M], *c=new int[M];
   int *dev_a, *dev_b, *dev_c;
   float tiempo1, tiempo2;
   hipEvent_t inicio1, fin1, inicio2, fin2; // para medir tiempos como con timestamp

   // fill the arrays 'a' and 'b' on the CPU
   for (int i=0; i<M; i++)
      a[i] = b[i] = i+1;

   hipEventCreate(&inicio1); // Se inicializan
   hipEventCreate(&fin1);
   hipEventRecord( inicio1, 0 ); // Se toma el tiempo de inicio

   d = M / N;
   printf( "d:%d\n", d );

   // allocate the memory on the GPU
   hipMalloc( (void**)&dev_a, M * sizeof(int) );
   hipMalloc( (void**)&dev_b, M * sizeof(int) );
   hipMalloc( (void**)&dev_c, M * sizeof(int) );

   // copy the arrays 'a' and 'b' to the GPU
   hipMemcpy( dev_a, a, M * sizeof(int), hipMemcpyHostToDevice );
   hipMemcpy( dev_b, b, M * sizeof(int), hipMemcpyHostToDevice );

   hipEventCreate(&inicio2); // Se inicializan
   hipEventCreate(&fin2);
   hipEventRecord( inicio2, 0 ); // Se toma el tiempo de inicio

   for (int i=0; i<d; i++)
      add<<<N,1>>>( dev_a, dev_b, dev_c, i );

   hipEventRecord( fin2, 0); // Se toma el tiempo final.
   hipEventSynchronize( fin2 ); // Se sincroniza
   hipEventElapsedTime( &tiempo2, inicio2, fin2 );

   // copy the array 'c' back from the GPU to the CPU
   hipMemcpy( c, dev_c, M * sizeof(int), hipMemcpyDeviceToHost );

   // free the memory allocated on the GPU
   hipFree( dev_a );
   hipFree( dev_b );
   hipFree( dev_c );

   hipEventRecord( fin1, 0); // Se toma el tiempo final.
   hipEventSynchronize( fin1 ); // Se sincroniza
   hipEventElapsedTime( &tiempo1, inicio1, fin1 );

   // display the results
   for (int i=0; i<M; i++)
       printf( "%d + %d = %d\n", a[i], b[i], c[i] );

   free(a);
   free(b);
   free(c);

   printf("Tiempo cálculo %f ms\n", tiempo2);
   printf("Tiempo total %f ms\n", tiempo1);

   return 0;
}
