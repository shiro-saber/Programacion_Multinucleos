/*
 * Copyright 1993-2010 NVIDIA Corporation.  All rights reserved.
 *
 * NVIDIA Corporation and its licensors retain all intellectual property and 
 * proprietary rights in and to this software and related documentation. 
 * Any use, reproduction, disclosure, or distribution of this software 
 * and related documentation without an express license agreement from
 * NVIDIA Corporation is strictly prohibited.
 *
 * Please refer to the applicable NVIDIA end user license agreement (EULA) 
 * associated with this source code for terms and conditions that govern 
 * your use of this NVIDIA software.
 * 
 */



#include <hip/hip_runtime.h>
#include <stdio.h>
//GAHGHAGHAGHAHGAHGHA
// N= bloques
 // M= calculos 
#define N   1000
#define M   1000000
int d;

__global__ void add( int *a, int *b, int *c, int i ) {
   int tid = i*N+blockIdx.x; // vector index
   if (tid < M)
      c[tid] = a[tid] + b[tid];
}//funcion de kernel cuda

int main( void ) {
   int *a= new int[M], *b=new int[M], *c=new int[M];
   int *dev_a, *dev_b, *dev_c;
   float tiempo1, tiempo2;
   hipEvent_t inicio1, fin1, inicio2, fin2; // para medir tiempos como con timestamp

   // llenar el arreglo NO en apralelo
   for (int i=0; i<M; i++)
      a[i] = b[i] = i+1;

   hipEventCreate(&inicio1); // Se inicializan
   hipEventCreate(&fin1); // un comentario por aqui 
   hipEventRecord( inicio1, 0 ); // Se toma el tiempo de inicio

   d = M / N;
   //printf( "d:%d\n", d );

   // allocate the memory on the GPU
   hipMalloc( (void**)&dev_a, M * sizeof(int) );
   hipMalloc( (void**)&dev_b, M * sizeof(int) );
   hipMalloc( (void**)&dev_c, M * sizeof(int) );//memoria en cuda 

   // copy the arrays 'a' and 'b' to the GPU
   hipMemcpy( dev_a, a, M * sizeof(int), hipMemcpyHostToDevice );// pasamos los datos a las GPU
   hipMemcpy( dev_b, b, M * sizeof(int), hipMemcpyHostToDevice );

   hipEventCreate(&inicio2); // Se inicializan
   hipEventCreate(&fin2);// otro comentario casual 
   hipEventRecord( inicio2, 0 ); // Se toma el tiempo de inicio

   for (int i=0; i<d; i++)
      add<<<N,1>>>( dev_a, dev_b, dev_c, i );
//llamar la funcion del kernel de cuda 
   hipEventRecord( fin2, 0); // Se toma el tiempo final.
   hipEventSynchronize( fin2 ); // Se sincroniza
   hipEventElapsedTime( &tiempo2, inicio2, fin2 );//un comentario temporal 

   // copy the array 'c' back from the GPU to the CPU
   hipMemcpy( c, dev_c, M * sizeof(int), hipMemcpyDeviceToHost );//traduccion: regresamos los datos a ram

   // free the memory allocated on the GPU
   hipFree( dev_a );
   hipFree( dev_b );
   hipFree( dev_c );//limpiamos la memoria de cuda

   hipEventRecord( fin1, 0); // Se toma el tiempo final.
   hipEventSynchronize( fin1 ); // Se sincroniza
   hipEventElapsedTime( &tiempo1, inicio1, fin1 );// otro comentario temporal

   // display the results
   //for (int i=0; i<M; i++)
   //    printf( "%d + %d = %d\n", a[i], b[i], c[i] );

   free(a);//a es libre 
   free(b);// b es un elfo libre
   free(c);//sin patria ni libertad estaremos muy lejos de la verdadera felicidad --atte: c

   printf("tiempo calculos en ms: %f\t tiempo de total %f\n", tiempo2,tiempo1);// asi o mas comentado?
   return 0;
}
