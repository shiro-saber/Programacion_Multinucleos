/*
 * Copyright 1993-2010 NVIDIA Corporation.  All rights reserved.
 *
 * NVIDIA Corporation and its licensors retain all intellectual property and 
 * proprietary rights in and to this software and related documentation. 
 * Any use, reproduction, disclosure, or distribution of this software 
 * and related documentation without an express license agreement from
 * NVIDIA Corporation is strictly prohibited.
 *
 * Please refer to the applicable NVIDIA end user license agreement (EULA) 
 * associated with this source code for terms and conditions that govern 
 * your use of this NVIDIA software.
 * 
 */

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <omp.h>

#define N   1000000

void add( int *a, int *b, int *c ) {
   int tid = 0;    // this is CPU zero, so we start at zero
   while (tid < N) {
      c[tid] = a[tid] + b[tid];
      tid ++;   // we have one CPU, so we increment by one
   }
}

int main( void ) {
   int *a=new int[N], *b = new int[N], *c = new int[N];
   hipEvent_t inicio, fin,st2,fn2;
   float tiempo, tt2;
	
   hipEventCreate( &st2 );
   hipEventCreate( &fn2 );
   // fill the arrays 'a' and 'b' on the CPU
   for (int i=0; i<N; i++)
      a[i] = b[i] = i+1;

   hipEventCreate( &inicio );
   hipEventCreate( &fin );
   hipEventRecord( inicio, 0 );
   add( a, b, c );
   hipEventRecord( fin, 0 );
   hipEventSynchronize( fin );
   hipEventElapsedTime( &tiempo, st2, fn2 );

   // display the results
   //for (int i=0; i<N; i++)
   //   printf( "%d + %d = %d\n", a[i], b[i], c[i] );

   free(a);
   free(b);
   free(c);
   hipEventRecord( fn2, 0 );
   hipEventSynchronize( fn2 );
   hipEventElapsedTime( &tt2, inicio, fin );
   printf("tiempo total en ms: %f\n", tiempo);

   return 0;
}
