/*
 * Copyright 1993-2010 NVIDIA Corporation.  All rights reserved.
 *
 * NVIDIA Corporation and its licensors retain all intellectual property and
 * proprietary rights in and to this software and related documentation.
 * Any use, reproduction, disclosure, or distribution of this software
 * and related documentation without an express license agreement from
 * NVIDIA Corporation is strictly prohibited.
 *
 * Please refer to the applicable NVIDIA end user license agreement (EULA)
 * associated with this source code for terms and conditions that govern
 * your use of this NVIDIA software.
 *
 */

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <omp.h>

#define N 1000000

void add( int *a, int *b, int *c ) {
   int tid = 0;    
   while (tid < N) {
      c[tid] = a[tid] + b[tid];
      tid ++;   
   }//#noSeHacerUnFor
}// y tal vez algunos calculos 

int main( void ) {
   int *a=new int[N], *b = new int[N], *c = new int[N];
   hipEvent_t inicio, fin,st2,fn2;
   float tiempo, tt2;

   hipEventCreate( &st2 );
   hipEventCreate( &fn2 );
   hipEventRecord( st2, 0 );
   // para medir el tiempo TOTAL
   for (int i=0; i<N; i++)
      a[i] = b[i] = i+1;//FOR para dar valores iniciales

   hipEventCreate( &inicio );
   hipEventCreate( &fin );
   hipEventRecord( inicio, 0 );//para medir el tiempo de EJECUCION
   add( a, b, c );
   hipEventRecord( fin, 0 );
   hipEventSynchronize( fin );
   hipEventElapsedTime( &tiempo, inicio, fin );//paramos cronometro y medimos tiempo

   // display the results
   //for (int i=0; i<N; i++)
   //   printf( "%d + %d = %d\n", a[i], b[i], c[i] );
   //for para testear

   free(a);
   free(b);
   free(c);
   hipEventRecord( fn2, 0 );
   hipEventSynchronize( fn2 );
   hipEventElapsedTime( &tt2, st2, fn2 );//paramos cronometro y medimos tiempo
   printf("tiempo calculos en ms: %f\t tiempo de total %f\n", tiempo,tt2);//cazar elefantes... o imprimir no se solo soy un comentario

   return 0;
}
