#include "hip/hip_runtime.h"
#include <stdio.h>
#include "book.h"

/* experiment with N */
/* how large can it be? 536870911 cm */
#define imin(a,b) (a<b?a:b)
#define N 1000000
const int THREADS_PER_BLOCK = 256;
const int blocksPerGrid =
            imin( 32, (N/2+THREADS_PER_BLOCK-1) / THREADS_PER_BLOCK );

__global__ void add(int size, int *a, int *b, int *c)
{
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  if (index < size)
    c[index] = a[index] + b[index];
  //__shared__ float cache[THREADS_PER_BLOCK];
  //int tid = threadIdx.x + blockIdx.x * blockDim.x;
  //int cacheIndex = threadIdx.x;

  //float   temp = 0;
  //while (tid < size) {
  //  c[tid] = a[tid] * b[tid];
    //  tid += blockDim.x * gridDim.x;
  //}
}//funcion de kernel cuda

struct DataStruct{
  int deviceID;
  int size;
  int *a;
  int *b;
  int *c;
};

void *addGPU(void *pvoidData)
{
   DataStruct *data = (DataStruct*)pvoidData;
   hipSetDevice(data->deviceID);

   int *a, *b, *c;
   int *d_a, *d_b, *d_c;
   int size = data->size;
   float tiempo1, tiempo2;
   hipEvent_t inicio1, fin1, inicio2, fin2; // para medir tiempos como con timestamp

   /* allocate space for host copies of a, b, c and setup input alues */
   a = data->a;
   b = data->b;
   c = data->c;

   hipEventCreate(&inicio1); // Se inicializan
   hipEventCreate(&fin1);
   hipEventRecord( inicio1, 0 ); // Se toma el tiempo de inicio

   /* allocate space for device copies of a, b, c */

   hipMalloc( (void **) &d_a, size);
   hipMalloc( (void **) &d_b, size);
   hipMalloc( (void **) &d_c, size);

   /* copy inputs to deice */
   /* fix the parameters needed to copy data to the device */
   hipMemcpy( d_a, a, size, hipMemcpyHostToDevice );// pasamos los datos a las GPU
   hipMemcpy( d_b, b, size, hipMemcpyHostToDevice );

   hipEventCreate(&inicio2); // Se inicializan
   hipEventCreate(&fin2);
   hipEventRecord( inicio2, 0 ); // Se toma el tiempo de inicio

   /* launch the kernel on the GPU */
   add<<< blocksPerGrid, THREADS_PER_BLOCK >>>(size, d_a, d_b, d_c );

   hipEventRecord( fin2, 0); // Se toma el tiempo final.
   hipEventSynchronize( fin2 ); // Se sincroniza
   hipEventElapsedTime( &tiempo2, inicio2, fin2 );

   /* copy result back to host */
   /* fix the parameters needed to copy data back to the host */
   hipMemcpy( c, d_c, size, hipMemcpyDeviceToHost );//traduccion: regresamos los datos a ram

   hipFree( d_a );
   hipFree( d_b );
   hipFree( d_c );//limpiamos la memoria de cuda

   hipEventRecord( fin1, 0); // Se toma el tiempo final.
   hipEventSynchronize( fin1 ); // Se sincroniza
   hipEventElapsedTime( &tiempo1, inicio1, fin1 );

   data->c = c;
   printf("tiempo calculos en ms: %f\t tiempo de total %f\n", tiempo2,tiempo1);

   return 0;
}

int main()
{
  hipDeviceProp_t prop;
  int videos = 0;
  hipGetDeviceCount(&videos);
  for (int i = 0; i < videos; ++i)
  {
    hipGetDeviceProperties(&prop, i);
    printf("%d -> # tarjetas \n", videos);
  }

  int *a = (int *)malloc(sizeof(int)*N);
  int *b = (int *)malloc(sizeof(int)*N);
  int *c = (int *)malloc(sizeof(int)*N);

  for( int i = 0; i < N; i++ )
  {
     a[i] = b[i] = i+1;
     c[i] = 0;
  }

  DataStruct data[2];
  data[0].deviceID = 0;
  data[0].size = N*(3/4);
  data[0].a = a;
  data[0].b = b;
  data[0].c = c;

  data[1].deviceID = 1;
  data[1].size = N/4;
  data[1].a = a - N/2;
  data[1].b = b - N/2;
  data[1].c = c - N/2;

  CUTThread thread = start_thread(addGPU, &(data[0]));
  addGPU(&(data[1]));
  end_thread(thread);

  //for (int i=0; i<N; i++)
  //   printf( "%d + %d = %d\n", a[i], b[i], c[i] );

  /* clean up */

  free(a);
  free(b);
  free(c);

  return 0;
}
