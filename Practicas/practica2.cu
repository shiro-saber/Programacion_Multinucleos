
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
//M*N

#define M 32

//llamada de kernel de cuda
__global__ void euler(int **a, int **b, int **c, int n, int m, int p)
{
  // aqui es cuando empezamos a llorar
  int i = blockIdx.x*blockDim.x + threadIdx.x;
  int j = blockIdx.y*blockDim.y + threadIdx.y;

  __syncthreads();
  c[i+j][j] = b[i+j][j] + a[i+j][j];
}

//funcion para imprimir
void print_matrix(int** mat, int kuz, int kuzemac)
{
  for(int i = 0; i < kuz; i++)
  {
      for(int j = 0; j < kuzemac; j++)
          printf(" %d |", mat[i][j]);

      printf("\n");
  }
}

//funcion que hara el traspaso a la memoria de videoy el regreso de la misma
float eulerGPU(int **a, int **b, int **c, int n, int m, int p)
{
  int **c_a, **c_b, **c_c;
  hipEvent_t inicio, fin;
  float tiempo_c;

  // reservamos memoria para cada parte de video
  hipMalloc((void**)&c_a, n*sizeof(int*));
  hipMalloc((void**)&c_c, n*sizeof(int*));
  hipMalloc((void**)&c_b, p*sizeof(int*));
  for (int i = 0; i < n; ++i)
  {
    hipMalloc((void**)&c_a[i], m*sizeof(int));
    hipMalloc((void**)&c_c[i], m*sizeof(int));
  }
  for (int j = 0; j < p; ++j)
    hipMalloc((void**)&c_b[j], p*sizeof(int));

  //copiamos a GPU
  hipMemcpy(c_a, a, n*m*sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(c_b, b, p*p*sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(c_c, c, n*m*sizeof(int), hipMemcpyHostToDevice);

  //definimos las dimensiones de los bloques y los threads
  dim3 bloques(p/M, p/M);
  dim3 threads(M, M);

  //tomar el tiempo \o/
  hipEventCreate(&inicio);
  hipEventCreate(&fin);
  hipEventRecord(inicio, 0);

  //llamas a cuda
  euler<<<bloques, threads>>>(c_a, c_b, c_c, n, m, p);

  //regresamos a CPU
  hipMemcpy(c, c_c, n*m*sizeof(int), hipMemcpyDeviceToHost);

  //terminmamos el tiempo
  hipEventRecord(fin, 0);
  hipEventSynchronize(fin);
  hipEventElapsedTime(&tiempo_c, inicio, fin);

  //imprimir los resultados
  print_matrix(c, n, m);

  return tiempo_c;
}

void inicializar(int** a, int** b, int** c_t, int n, int m, int p)
{
  hipEvent_t init, end;
  float tiempo_t;
  int d;
  int c = 255;

  //tiempos
  hipEventCreate(&init);
  hipEventCreate(&end);
  hipEventRecord(init, 0);

  // para inicializar la matriz
  for (int i = 0; i < n; ++i)
  {
    for (int j = 0; j < m; ++j)
      a[i][j] = c;
    c--;
  }

  for (int x = 0; x < n; ++x)
    for (int y = 0; y < m; ++y)
      c_t[x][y] = 0;

  for (int k = 0; k < p; ++k)
  {
    d = 1;
    for (int l = 0; l < p; ++l)
    {
      b[l][k] = d;
      d++;
    }
  }

  //neta teniamos que comentar cada linea?
  hipEventRecord(end, 0);
  hipEventSynchronize(end);
  hipEventElapsedTime(&tiempo_t, init, end);

  printf("El tiempo que tarda es: %fms en cálculos\nEl tiempo que tarda: %fms en total\n", eulerGPU(a,b,c_t,n,m,p), tiempo_t); //que? no llegaste? por eso el grito jajaja
}

void solve(char p, char n, char m)
{
  int **a, **b, **c;
  // a sera la matriz de n*m
  a = (int**)malloc(n*sizeof(int*));
  // b sera la matriz de p*p
  b = (int**)malloc(p*sizeof(int*));
  //c matriz resultante
  c = (int**)malloc(n*sizeof(int*));
  for (int i = 0; i < n; ++i)
  {
    a[i] = (int*)malloc(m*sizeof(int));
    c[i] = (int*)malloc(m*sizeof(int));
  }

  for (int j = 0; j < p; ++j)
    b[j] = (int*)malloc(p*sizeof(int));

  // inicializamos las matrices con los valores pedidos
  inicializar(a, b, c, n, m, p);
  free(a);
  free(b);
  free(c);
}

int main(int argc, char *argv[])
{
  if ( argc != 4 )
  {
     //printf("%d\n", argc);
     printf("%s 1- valor de p 2- valor de M 3- valor de N\n", argv[0]);
     exit(0);
  }
  if (atoi(argv[1]) % 2 == 0 && atoi(argv[1]) > 7)
  {
    printf("P tiene que ser impar y se permite máximo 7x7\n");
    exit(0);
  }

  solve(atoi(argv[1]), atoi(argv[2]), atoi(argv[3]));
  // Ya di el grito
  return 0;
}
