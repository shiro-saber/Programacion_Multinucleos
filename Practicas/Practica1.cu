/* primer practica Moi */

#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>
#include <math.h>

/* Definicion de bloques y threads por bloque */
#define N 1000000
#define THREADS_PER_BLOCK 1000
/* Números a evaluar */
#define max 1000000

//kernel de CUDA
__global__ void primos(int *n_c, int *raiz_c)
{
  //sacamos el index
  int index = blockIdx.x * blockDim.x + threadIdx.x;

  n_c[0] = 1;
  n_c[1] = 1;

  /*if(index < max)
  {
    for(int i = index; i < raiz_c[0]; i+=2)
      n_c[i] = 1;
  }*/

  if(index <= raiz_c[0])
  {
    //for(int j = index*index; j < max; j+=index) => esta era la buena
    for(int j = index; j < max; j += index)
      n_c[j] = 1;
  }
}

int main(void)
{
  int *raiz;
  int number = 0;
  //arreglo
  int *n;
  //arreglo de cuda
  int *n_c;
  int *raiz_c;
  //tamaño del arreglo
  int size = max*sizeof(int);
  /* para tomar los tiempos */
  float tiempo1, tiempo2;
  hipEvent_t inicio1, fin1, inicio2, fin2; // para medir tiempos como con timestamp

  /* Inicializacion de tiempo de ejecución */
  hipEventCreate(&inicio1); // Se inicializan
  hipEventCreate(&fin1);
  hipEventRecord( inicio1, 0 ); // Se toma el tiempo de inicio

  //asignacion de memoria CPU
  n = (int*)malloc(sizeof(int)*max);
  raiz = (int*)malloc(sizeof(int)*1);

  //llenamos el arreglo en 0
  for(int i = 0; i < max; i++)
    n[i] = 0;

  raiz[0] = sqrt(max);

  //asignacion de memoria de cuda
  hipMalloc((void **) &n_c, size);
  hipMalloc((void **) &raiz_c, 1);
  //copiamos el arreglo a GPU
  hipMemcpy(n_c, n, size, hipMemcpyHostToDevice);
  hipMemcpy(raiz_c, raiz, 1, hipMemcpyHostToDevice);

  /* Tiempos de ejecución */
  hipEventCreate(&inicio2); // Se inicializan
  hipEventCreate(&fin2);
  hipEventRecord(inicio2, 0); // Se toma el tiempo de inicio
  //llama del kernel de CUDA
  primos<<< N/THREADS_PER_BLOCK, THREADS_PER_BLOCK >>>(n_c, raiz_c);
  /* Paramos el crono de ejecución */
  hipEventRecord(fin2,0);
  hipEventSynchronize(fin2);
  hipEventElapsedTime(&tiempo2, inicio2, fin2);

  //Regresamos el arreglo a CPU
  hipMemcpy(n, n_c, size, hipMemcpyDeviceToHost);

  for(int j = 0; j < max; j++)
    if(n[j] == 0)
    {
      //printf("%d\t", j);
      number++;
    }

  //Liberamos memoria en GPU
  hipFree(n_c);
  hipFree(raiz_c);

  printf("El número de números primos en %d es: %d\n", max, number);
  printf("Herr Moy\n");
  hipEventRecord(fin1, 0); // Se toma el tiempo final.
  hipEventSynchronize(fin1); // Se sincroniza
  hipEventElapsedTime(&tiempo1, inicio1, fin1);

  printf("El tiempo de ejecución es: %f\tEl tiempo de cálculos es: %f\n", tiempo1, tiempo2);

  //liberamos memoria de CPU
  free(n);
  free(raiz);
  return 0;
}
