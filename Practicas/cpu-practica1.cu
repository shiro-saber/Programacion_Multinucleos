
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>

// Algoritmo Criba de Eratóstenes
void primos(unsigned long max)
{
   unsigned long i, j, c=0;
   max++;
   char *arr = new char[max];
   hipEvent_t inicio, fin;
   float tiempo;

   hipEventCreate( &inicio );
   hipEventCreate( &fin );
   hipEventRecord( inicio, 0 );

   if (max >= 2)
   {
      for (i=0; i<max; i++)
         arr[i] = 0;
      arr[0] = 1;
      arr[1] = 1;

      unsigned long raiz = sqrt(max);

      for (j=4; j<max; j+=2)
         arr[j] = 1;

      for (i=3; i<=raiz; i+=2) // impares
         if (arr[i] == 0)
            for (j=i*i; j<max; j+=i)
               arr[j] = 1;

      hipEventRecord( fin, 0 );
      hipEventSynchronize( fin );
      hipEventElapsedTime( &tiempo, inicio, fin );

      for (i=0; i<max; i++)
         if (arr[i] == 0)
         {
//            printf("%ld ", i);
            c++;
         }
      printf("\n total:%ld\n", c);
   }
   free(arr);
   printf("tiempo total en ms: %f\n", tiempo);
}

int main(int argc, char *argv[])
{
   primos(100000000);
   return 1;
}
