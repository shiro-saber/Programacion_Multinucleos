#include "hip/hip_runtime.h"
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/sequence.h>
#include <thrust/functional.h>
#include <thrust/sort.h>
#include <thrust/transform.h>
#include <chrono>
#include <algorithm>
#include <vector>

using namespace thrust;
using namespace std;

using sys_clock = std::chrono::system_clock;

void thrust_sequence(){
  thrust::device_vector<int> D_vec(10,1);
  thrust::fill(D_vec.begin(),D_vec.begin() + 7,9);

  thrust::host_vector<int> H_vec(D_vec.begin(),D_vec.begin() +5 );

  thrust::sequence(H_vec.begin(),H_vec.end(),5,2);
  thrust::copy(H_vec.begin(),H_vec.end(),D_vec.begin());
  int i=0;
  for (auto value: D_vec)
    std::cout<< "d[" << i++ << "]" << value << std::endl;
}

void sorts(){
  int current_h =0, current_d =0, exit =0, limit = 1 << 24;
  chrono::time_point<sys_clock> t1,t2;
  chrono::duration<double,milli> exec_time_ms;

  host_vector<int> H_vec(limit);
  thrust::generate(H_vec.begin(),H_vec.end(),rand);
    device_vector<int> D_vec = H_vec;

    t1 = sys_clock::now();
    thrust::sort(D_vec.begin(), D_vec.end());
    t2 = sys_clock::now();
    exec_time_ms = t2-t1;

    cout << "gpu sort time : " << exec_time_ms.count() << endl;
    vector<int> st1_hsot_vec(H_vec.size());
    thrust::copy(H_vec.begin(), H_vec.end(), st1_hsot_vec.begin());

    t1 = sys_clock::now();
    std::sort(st1_hsot_vec.begin(),st1_hsot_vec.end());
    t2 = sys_clock::now();
    exec_time_ms = t2-t1;
    cout<< "CPU time: " << exec_time_ms.count() << endl;
}
struct functor
{
  const float a;
  functor(float _a):a(_a){}
  __host__ __device__ float operator()(const float &x, const float &y)const{return a*x +y;}
};

void transforms(){
  const float A=5;
  const int size = 10;

  host_vector<float> X(size), Y(size);
  sequence(X.begin(),X.end(),10,10);
  sequence(Y.begin(),Y.end(),10,10);

  thrust::transform(X.begin(),X.end(),Y.begin(),Y.end(),functor(A));

  for(int i =0; i < Y.size(); i++){
    cout << "Y[" << i << "]=" << Y[i] << endl;
  }

}



template <typename T>
struct square
{

  __host__ __device__ float operator()(const T &x)const{return x*x;}
};

int main (void){

  float x[4] = {1,2,3,4};

  device_vector<float> D_vec(x,x+4);
  square<float> unary_op;
  thrust::plus<float> binary_op;
  float norm= std::sqrt(
    thrust::transform_reduce(D_vec.begin(),D_vec.end(),unary_op,0,binary_op)
  );

  cout << norm << endl;
  return 0;
}
// nvcc thrust.cu -std=c++11 -D_MWAITXINTRIN_H_INCLUDED
