#include "hip/hip_runtime.h"
// confio en la obviedad de esto
#include <math.h>
#include <string.h>
#include <openacc.h>
#include "timer.h"

int main(int argc, char** argv)
{
  int n = 4096; //matriz en n
  int m = 4096; //matriz en m
  int iter_max = 1000; //numero maximo de intentos para el resultado

  const float pi  = 2.0f * asinf(1.0f); //viva, es el valor de pi
  const float tol = 1.0e-5f; //es una variable del metodo de Laplace
  float error     = 1.0f; //el error permitido

  float A[n][m]; //matriz n*m
  float Anew[n][m]; //la que sera de cuda
  float y0[n]; //elementps

  memset(A, 0, n * m * sizeof(float)); //reservamos memoria

  // Ponemos las barreras
  for (int i = 0; i < m; i++)
  {
    A[0][i]   = 0.f; //en la barrera es 0
    A[n-1][i] = 0.f; //en la barrera es 0
  }

  for (int j = 0; j < n; j++)
  {
    y0[j] = sinf(pi * j / (n-1)); //una aplicacion del seno que para algo ha de servir
    A[j][0] = y0[j]; //elemento
    A[j][m-1] = y0[j]*expf(-pi); //algo mas del metodo de laplace
  }

  #if _OPENACC
    acc_init(acc_device_nvidia); //inciamos el ambiente de openAcc
  #endif

  printf("Jacobi relaxation Calculation: %d x %d mesh\n", n, m); //print

  StartTimer(); //esta tomando el tiempo
  int iter = 0; //en que iteracion vamos.

  #pragma omp parallel for shared(Anew) // en cpu
  for (int i = 1; i < m; i++)
  {
    Anew[0][i]   = 0.f;
    Anew[n-1][i] = 0.f;
  }

  #pragma omp parallel for shared(Anew) //en cpu
  for (int j = 1; j < n; j++)
  {
    Anew[j][0]   = y0[j];
    Anew[j][m-1] = y0[j]*expf(-pi); //ya lo hizo varias veces, lo importante es acc
  }

  #pragma acc data copy(A), create(Anew) //dejamos que pragma acc copie a GPU
  while (error > tol && iter < iter_max) //hasta que no lleguemos al resultado y a la ultima iteracion
  {
    error = 0.f;
    #pragma omp parallel for shared(m, n, Anew, A) //paralelizamos en cpu el envio de datos a GPU
      #pragma acc kernels loop gang(32), vector(16) //paralelizamos los kernels, los for y asignamos bloques y threads
        for( int j = 1; j < n-1; j++)
        {
          #pragma acc loop gang(16), vector(32) //paralelizamos otro loop con bloques y threads
          for( int i = 1; i < m-1; i++ )
          {
            Anew[j][i] = 0.25f * ( A[j][i+1] + A[j][i-1] + A[j-1][i] + A[j+1][i]); //comentario
            error = fmaxf( error, fabsf(Anew[j][i]-A[j][i])); //algo de Jacobi creo
          }
        }

  #pragma omp parallel for shared(m, n, Anew, A) //de nueva cuenta paralelizamos el envio de datos desde CPU
    #pragma acc kernels loop //de nueva cuenta paralelizamos los kernels que van a trabajar y los fot
    for( int j = 1; j < n-1; j++)
    {
      #pragma acc loop gang(16), vector(32) //mas for
      for( int i = 1; i < m-1; i++ )
      {
        A[j][i] = Anew[j][i]; // yeii
      }
    }

    if(iter % 100 == 0) printf("%5d, %0.6f\n", iter, error); //no se encontro
    iter++; //++ ++ ++ ++ ++ ++
  }

  double runtime = GetTimer(); //tiempo
  printf(" total: %f s\n", runtime / 1000.f); //me tarde
}
