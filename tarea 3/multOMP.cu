
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>

int i, j, k;

//funcion que llena la matriz randonmente
float** rand_matrix(float **mat, int kuz)
{
  float **backup = mat;

  for(i = 0; i < kuz; i++)
    for(j = 0; j < kuz; j++)
      mat[i][j] = (rand()%1000+1);

    return backup;
}

//funcion para imprimir la matriz
void print_matrix(float **mat, int kuz)
{
    for(i = 0; i < kuz; i++)
    {
        for(j = 0; j < kuz; j++)
            printf(" %f |", mat[i][j]);

        printf("\n");
    }
}

int main(int argc, char *argv[])
{
  if (argc != 3)
  {
    fprintf(stderr, "The right use it's %s <number of the N*N matrix> <0 if you want prints or 1 if you don't>\n", argv[0]);
    exit(-1);
  }

  hipEvent_t inicio, fin, st2, fn2;
  float tiempo, tt2;
  float **mat, **mat2, **res;
  int N = atoi(argv[1]);

  //iniciamos el crono
  hipEventCreate( &st2 );
  hipEventCreate( &fn2 );
  hipEventRecord( st2, 0 );

  // aloja memoria para la matriz
  mat = (float **)malloc(N*sizeof(float*));
  mat2 = (float **)malloc(N*sizeof(float*));
  res = (float **)malloc(N*sizeof(float*));
  for(i = 0; i < N; i++)
  {
    //aloja la memoria por celda de la matriz
    mat[i] = (float *)malloc(N*sizeof(float));
    mat2[i] = (float *)malloc(N*sizeof(float));
    res[i] = (float *)malloc(N*sizeof(float));
  }

  //llena las matrices
  mat = rand_matrix(mat, N);
  mat2 = rand_matrix(mat2, N);

  //imprime las matrices
  if(atoi(argv[2]) == 0)
  {
    print_matrix(mat, N);
    printf("\n\n\n");
    print_matrix(mat2, N);
  }

  //crono de los calculos
  hipEventCreate( &inicio );
  hipEventCreate( &fin );
  hipEventRecord( inicio, 0 );

  //hacemos la multiplicacion
  #pragma omp parallel for private(k)
    for(i=0; i<N; ++i)
      for(j=0; j<N; ++j)
        for(k=0; k<N; ++k)
          res[i][j]+=mat[i][k]*mat2[k][j];

  //paramos el crono de los calculos
  hipEventRecord( fin, 0 );
  hipEventSynchronize( fin );
  hipEventElapsedTime( &tiempo, inicio, fin );

  //imprimimos el resultado
  if(atoi(argv[2]) == 0)
  {
    printf("\n\n\n");
    print_matrix(res,N);
  }

  //liberamesta
  free(mat);
  free(mat2);
  free(res);

  hipEventRecord( fn2, 0 );
  hipEventSynchronize( fn2 );
  hipEventElapsedTime( &tt2, st2, fn2 );//paramos cronometro y medimos tiempo total
  printf("tiempo calculos en ms: %f\t tiempo de total %f\n", tiempo,tt2);//cazar elefantes... o imprimir no se solo soy un comentario
}
